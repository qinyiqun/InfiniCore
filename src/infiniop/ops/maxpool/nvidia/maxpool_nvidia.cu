#include "../../../devices/nvidia/nvidia_common.cuh"
#include "../../../devices/nvidia/nvidia_handle.cuh"
#include "maxpool_nvidia.cuh"

#define DESTROY_CUDNN_DESCRIPTOR(desc_ptr, destroy_func) \
    do {                                                 \
        if (desc_ptr) {                                  \
            destroy_func(desc_ptr);                      \
            desc_ptr = nullptr;                          \
        }                                                \
    } while (0)

#define CLEANUP_CUDNN_DESCRIPTORS()                                            \
    do {                                                                       \
        DESTROY_CUDNN_DESCRIPTOR(input_desc, hipdnnDestroyTensorDescriptor);    \
        DESTROY_CUDNN_DESCRIPTOR(output_desc, hipdnnDestroyTensorDescriptor);   \
        DESTROY_CUDNN_DESCRIPTOR(pooling_desc, hipdnnDestroyPoolingDescriptor); \
    } while (0)

namespace op::maxpool::nvidia {

struct Descriptor::Opaque {
    std::shared_ptr<device::nvidia::Handle::Internal> internal;
    size_t workspace_size = 0;

#ifdef ENABLE_CUDNN_API
    hipdnnTensorDescriptor_t input_desc = nullptr;
    hipdnnTensorDescriptor_t output_desc = nullptr;
    hipdnnPoolingDescriptor_t pooling_desc = nullptr;
#endif

private:
    Opaque(std::shared_ptr<device::nvidia::Handle::Internal> internal_ptr)
        : internal(internal_ptr) {}

#ifdef ENABLE_CUDNN_API
    infiniStatus_t getCudnnDataType(infiniDtype_t data_type,
                                    hipdnnDataType_t &cudnn_data_type) const {
        if (data_type == INFINI_DTYPE_F16) {
            cudnn_data_type = device::nvidia::getCudnnDtype(data_type);
        } else if (data_type == INFINI_DTYPE_F32) {
            cudnn_data_type = device::nvidia::getCudnnDtype(data_type);
        } else if (data_type == INFINI_DTYPE_BF16) {
            cudnn_data_type = device::nvidia::getCudnnDtype(data_type);
        } else {
            return INFINI_STATUS_BAD_TENSOR_DTYPE;
        }
        return INFINI_STATUS_SUCCESS;
    }

    infiniStatus_t createPoolingDescriptors(const MaxPoolInfo &info,
                                            hipdnnDataType_t cudnn_data_type) {
        // Create CUDNN descriptors
        CHECK_CUDNN(hipdnnCreateTensorDescriptor(&input_desc));
        CHECK_CUDNN(hipdnnCreateTensorDescriptor(&output_desc));
        CHECK_CUDNN(hipdnnCreatePoolingDescriptor(&pooling_desc));

        // Setup tensor descriptors
        std::vector<int> input_dims_vec = {static_cast<int>(info.batch),
                                           static_cast<int>(info.channels)};
        std::vector<int> output_dims_vec = {static_cast<int>(info.batch),
                                            static_cast<int>(info.channels)};

        for (size_t i = 0; i < info.ndim; ++i) {
            input_dims_vec.push_back(static_cast<int>(info.input_dims[i]));
            output_dims_vec.push_back(static_cast<int>(info.output_dims[i]));
        }

        if (info.ndim == 1) {
            // For 1D pooling, add dummy dimension
            input_dims_vec.push_back(1);
            output_dims_vec.push_back(1);
        }

        CHECK_CUDNN(cudnnSetTensorNdDescriptorEx(
            input_desc, HIPDNN_TENSOR_NCHW, cudnn_data_type, input_dims_vec.size(),
            input_dims_vec.data()));

        CHECK_CUDNN(cudnnSetTensorNdDescriptorEx(
            output_desc, HIPDNN_TENSOR_NCHW, cudnn_data_type, output_dims_vec.size(),
            output_dims_vec.data()));

        return INFINI_STATUS_SUCCESS;
    }

    infiniStatus_t setupPoolingDescriptor(const MaxPoolInfo &info) {
        // Setup pooling descriptor
        std::vector<int> kernel_vec, stride_vec, pad_vec;
        for (size_t i = 0; i < info.ndim; ++i) {
            kernel_vec.push_back(static_cast<int>(info.kernel_sizes[i]));
            stride_vec.push_back(static_cast<int>(info.strides[i]));
            pad_vec.push_back(static_cast<int>(info.pads[i]));
        }

        if (info.ndim == 1) {
            // For 1D pooling, add dummy dimension
            kernel_vec.push_back(1);
            stride_vec.push_back(1);
            pad_vec.push_back(0);
        }

        CHECK_CUDNN(hipdnnSetPoolingNdDescriptor(
            pooling_desc, HIPDNN_POOLING_MAX, HIPDNN_NOT_PROPAGATE_NAN,
            kernel_vec.size(), kernel_vec.data(), pad_vec.data(),
            stride_vec.data()));

        return INFINI_STATUS_SUCCESS;
    }

    infiniStatus_t initializeCudnnContext(MaxPoolInfo &info,
                                          infiniDtype_t data_type) {
        hipdnnDataType_t cudnn_data_type;
        CHECK_STATUS(getCudnnDataType(data_type, cudnn_data_type));

        CHECK_STATUS(createPoolingDescriptors(info, cudnn_data_type));
        CHECK_STATUS(setupPoolingDescriptor(info));

        // Max pooling typically doesn't need workspace
        workspace_size = 0;

        return INFINI_STATUS_SUCCESS;
    }
#endif

public:
    Opaque(Opaque &&other) noexcept
        : internal(std::move(other.internal)),
          workspace_size(other.workspace_size)
    // clang-format off
#ifdef ENABLE_CUDNN_API
          , input_desc(other.input_desc)
          , output_desc(other.output_desc)
          , pooling_desc(other.pooling_desc)
#endif
    // clang-format on
    {
#ifdef ENABLE_CUDNN_API
        other.input_desc = nullptr;
        other.output_desc = nullptr;
        other.pooling_desc = nullptr;
#endif
        other.workspace_size = 0;
    }

    ~Opaque() {
#ifdef ENABLE_CUDNN_API
        CLEANUP_CUDNN_DESCRIPTORS();
#endif
    }

    static inline utils::Result<Opaque>
    create(std::shared_ptr<device::nvidia::Handle::Internal> internal_ptr,
           MaxPoolInfo &info, infiniDtype_t data_type) {
#ifdef ENABLE_CUDNN_API
        Opaque opaque(internal_ptr);
        auto status = opaque.initializeCudnnContext(info, data_type);
        if (status != INFINI_STATUS_SUCCESS) {
            return status;
        }
        return utils::Result<Opaque>(std::move(opaque));
#else
        return INFINI_STATUS_NOT_IMPLEMENTED;
#endif
    }
};

Descriptor::~Descriptor() {
    if (_opaque) {
        delete _opaque;
    }
}

infiniStatus_t Descriptor::create(infiniopHandle_t handle_,
                                  Descriptor **desc_ptr,
                                  infiniopTensorDescriptor_t output_desc,
                                  infiniopTensorDescriptor_t input_desc,
                                  void *kernel_size, void *strides, void *pads,
                                  bool ceil_mode) {

#ifdef ENABLE_CUDNN_API
    auto handle = reinterpret_cast<device::nvidia::Handle *>(handle_);
    auto dtype = input_desc->dtype();

    CHECK_DTYPE(dtype, INFINI_DTYPE_F16, INFINI_DTYPE_F32, INFINI_DTYPE_BF16);

    auto result = MaxPoolInfo::create(output_desc, input_desc, kernel_size,
                                      strides, pads, ceil_mode);
    CHECK_RESULT(result);
    auto info = result.take();

    auto opaque_result = Opaque::create(handle->internal(), info, dtype);
    CHECK_RESULT(opaque_result);
    auto opaque = new Opaque(opaque_result.take());

    *desc_ptr = new Descriptor(dtype, std::move(info), opaque->workspace_size,
                               opaque, handle->device, handle->device_id);

    return INFINI_STATUS_SUCCESS;
#else
    return INFINI_STATUS_NOT_IMPLEMENTED;
#endif
}

infiniStatus_t Descriptor::calculate(void *workspace, size_t workspace_size,
                                     void *output, const void *input,
                                     void *stream) const {

#ifdef ENABLE_CUDNN_API
    const float alpha = 1.0f, beta = 0.0f;

    // 打印input展平后的前十个数据
    //  printf("MaxPool input (first 10 elements): ");
    //  const uint16_t *input_data = static_cast<const uint16_t *>(input);
    //  for (int i = 0; i < 10; ++i) {
    //    // 将BF16转换为float显示
    //    union {
    //      uint32_t bits;
    //      float value;
    //    } converter;
    //    uint16_t bf16_val = input_data[i];
    //    converter.bits = static_cast<uint32_t>(bf16_val) << 16;
    //    printf("%f ", converter.value);
    //  }
    //  printf("\n");

    CHECK_STATUS(_opaque->internal->useCudnn(
        (hipStream_t)stream, [&](hipdnnHandle_t handle) {
            CHECK_CUDNN(hipdnnPoolingForward(handle, _opaque->pooling_desc, &alpha,
                                            _opaque->input_desc, input, &beta,
                                            _opaque->output_desc, output));
            return INFINI_STATUS_SUCCESS;
        }));

    return INFINI_STATUS_SUCCESS;
#else
    return INFINI_STATUS_NOT_IMPLEMENTED;
#endif
}

} // namespace op::maxpool::nvidia
